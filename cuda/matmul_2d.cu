
#include <hip/hip_runtime.h>
#define BM 128
#define BN 128
#define BK 8
#define TM 8
#define TN 8

__global__ void matmul_2d_tiling(float* a, float* b, float* c, const int M, const int N, const int K) {
    const int c_row = blockIdx.y;
    const int c_col = blockIdx.x;

    const int thread_row = threadIdx.y;
    const int thread_col = threadIdx.x;
    const int idx = thread_row * blockDim.x + thread_col;

    __shared__ float a_local[BM * BK];
    __shared__ float b_local[BK * BN];

    a += c_row * BM * K;
    b += c_col * BN;
    c += c_row * BM * N + c_col * BN;

    float thread_results[TM * TN] = {0};
    float regM[TM] = {0};
    float regN[TN] = {0};

    const int a_inner_col = idx % (BK / 4);
    const int a_inner_row = idx / (BK / 4);

    const int b_inner_col = idx % (BN / 4);
    const int b_inner_row = idx / (BN / 4);

    for (int block_idx = 0; block_idx < K; block_idx += BK) {
        // load 4 floats into local memory (L1 SMEM) 
        // thats BM / number of threads per block
        // meaning 128 / 32 = 4
        float4 tmp = reinterpret_cast<float4 *>(&a[a_inner_row * K + a_inner_col * 4])[0];
        a_local[(a_inner_col * 4 + 0) * BM + a_inner_row] = tmp.x;  
        a_local[(a_inner_col * 4 + 1) * BM + a_inner_row] = tmp.y;
        a_local[(a_inner_col * 4 + 2) * BM + a_inner_row] = tmp.z;
        a_local[(a_inner_col * 4 + 3) * BM + a_inner_row] = tmp.w;

        reinterpret_cast<float4 *>(&b_local[b_inner_row * BN + b_inner_col * 4])[0] = reinterpret_cast<float4 *>(&b[b_inner_row * N + b_inner_col * 4])[0];
        __syncthreads();

        // move the tile sideways
        a += BK;
        // move the tile downwards
        b += BK * N;

        for (int dot_idx = 0; dot_idx < BK; dot_idx++) {
            for (int a_idx = 0; a_idx < TM; a_idx++) {
                regM[a_idx] = a_local[(dot_idx * BM) + thread_row * TM + a_idx];
            }
            for (int b_idx = 0; b_idx < TN; b_idx++) {
                regN[b_idx] = b_local[(dot_idx * BN) + thread_col * TN + b_idx];
            }
            
            for (int res_idx_m = 0; res_idx_m < TM; res_idx_m++) {
                for (int res_idx_n = 0; res_idx_n < TN; res_idx_n++) {
                    thread_results[res_idx_m * TN + res_idx_n] += regM[res_idx_m] * regN[res_idx_n];
                }
            }
        }
        __syncthreads();
    }
    
    for (int res_idx_m = 0; res_idx_m < TM; res_idx_m++) {
        for (int res_idx_n = 0; res_idx_n < TN; res_idx_n++) {
            c[(thread_row * TM + res_idx_m) * N + thread_col * TN + res_idx_n] = thread_results[res_idx_m * TN + res_idx_n];
        }
    }
}