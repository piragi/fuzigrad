#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <cmath>
#include "constants.h"
#include "matmul_2d.h"
#include <iostream>
#include <cmath>
#include <map>

void matmul_cuda(float* h_a, float* h_b, float* h_c, int M, int N, int K) {
    float* d_a, * d_b, * d_c;
    // (BM * BN) / (WM * WN) * WARPSIZE = Number of threads per block
    int number_of_threads = 128;

    hipMalloc((void**)&d_a, sizeof(float) * M * K);
    hipMalloc((void**)&d_b, sizeof(float) * K * N);
    hipMalloc((void**)&d_c, sizeof(float) * M * N);

    hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    dim3 block(number_of_threads);
    dim3 grid(((M + (BM - 1)) / BM), ((N + (BN - 1)) / BN));

    matmul_2d_tiling << <grid, block >> > (d_a, d_b, d_c, M, N, K);

    hipMemcpy(h_c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void matmul_cublas(float* h_a, float* h_b, float* h_c_cublas, int M, int N, int K) {
    float* d_a, * d_b, * d_c;

    hipMalloc((void**)&d_a, sizeof(float) * M * K);
    hipMalloc((void**)&d_b, sizeof(float) * K * N);
    hipMalloc((void**)&d_c, sizeof(float) * M * N);

    hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_b, N, d_a, K, &beta, d_c, N);

    hipMemcpy(h_c_cublas, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void compare_arrays(float* a, float* b, int size) {
    std::map<float, int> error_counts; // To store unique errors and their counts

    for (int i = 0; i < size; i++) {
        double error = std::fabs(a[i] - b[i]);
        if (error > 0.0001) { // You may adjust the tolerance level as needed
            error_counts[error]++;
        }
    }

    // Print the unique errors and their counts
    printf("Unique Errors and Their Counts:\n");
    for (const auto& [error, count] : error_counts) {
        printf("Error: %f, Count: %d\n", error, count);
    }
}

int main() {
    const int M = 4096;
    const int N = 4096;
    const int K = 4096;

    float* h_a = (float*)malloc(sizeof(float) * M * K);
    float* h_b = (float*)malloc(sizeof(float) * K * N);
    float* h_c = (float*)malloc(sizeof(float) * M * N);
    float* h_c_cublas = (float*)malloc(sizeof(float) * M * N);

    srand(10);
    for (int i = 0; i < M * K; i++) h_a[i] = (float)(rand() % 9);
    for (int i = 0; i < K * N; i++) h_b[i] = (float)(rand() % 9);

    matmul_cuda(h_a, h_b, h_c, M, N, K);
    matmul_cublas(h_a, h_b, h_c_cublas, M, N, K);

    // print total error
    float total_error = 0.0f;
    for (int i = 0; i < M * N; i++) {
        total_error += std::abs(h_c[i] - h_c_cublas[i]);
    }
    printf("Total Error: %f\n", total_error);

    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_cublas);

    return 0;
}

