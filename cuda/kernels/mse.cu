#include "hip/hip_runtime.h"
#include <assert.h>
#include <cstdio>
#include "../constants.h"

__device__ void load_GMEM(float* a, float* b, const int M, const int N, float* a_local, float* b_local, const int inner_row, const int inner_col, const int stride) {
    for (int offset=0; offset < BM; offset+=stride) {
        reinterpret_cast<float4*>(&a_local[(inner_row + offset) * BK + inner_col * 4])[0] 
            = reinterpret_cast<float4*>(&a[(inner_row + offset) * N + inner_col * 4])[0];

        reinterpret_cast<float4*>(&b_local[(inner_row + offset) * BK + inner_col * 4])[0] 
            = reinterpret_cast<float4*>(&b[(inner_row + offset) * N + inner_col * 4])[0];
    }
}

__device__ void load_SMEM(float* a_local, float* b_local, const int M, const int N, float* reg_tile, const int inner_row, const int inner_col) {
    for (int i=0; i < MSE_TM; i++) {
        for (int j=0; j < MSE_TN; j++) {
            float a_el = a_local[(inner_row * MSE_TM + i) * BK + inner_col * MSE_TN + j];
            float b_el = b_local[(inner_row * MSE_TM + i) * BK + inner_col * MSE_TN + j];
            float difference =  a_el - b_el;
            *reg_tile += difference * difference;
        }
    }
}

extern "C" __global__ void mean_squared_error(float* a, float* b, float* block_result, const int M, const int N) {
    const int number_of_threads = blockDim.x * blockDim.y;
    const int idx = threadIdx.x;

    __shared__ float a_local[BM * BK];
    __shared__ float b_local[BM * BK];
    float reg_tile = 0.0;
    // when every thread accesses global memory does this make things slow?
    // should i fill up registers to the max and then fill up local memory and only then fill up block_result?

    // load into SMEM
    const int inner_row = idx / (BK / 4);
    const int inner_col = idx % (BK / 4);
    const int stride = (number_of_threads * 4) / BK;

    // bring a and b into position
    int position = blockIdx.x * BM * N + blockIdx.y * BK;
    a += position;
    b += position;
    block_result += blockIdx.x * gridDim.y + blockIdx.y;

    load_GMEM(a, b, M, N, a_local, b_local, inner_row, inner_col, stride);
    __syncthreads();
    load_SMEM(a_local, b_local, M, N, &reg_tile, inner_row, inner_col);
    __syncthreads();
    atomicAdd(block_result, reg_tile);
}