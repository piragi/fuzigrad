#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <assert.h>
#include <cmath>
#include "constants.h"
#include "matmul_2d.h"
#include <iostream>
#include <cmath>
#include <map>

void matmul_cuda(float* h_a, float* h_b, float* h_c, int M, int N, int K) {
    float* d_a, * d_b, * d_c;
    int number_of_threads = 256;

    hipMalloc((void**)&d_a, sizeof(float) * M * K);
    hipMalloc((void**)&d_b, sizeof(float) * K * N);
    hipMalloc((void**)&d_c, sizeof(float) * M * N);

    hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    dim3 block(BM / TM, BN / TN);
    dim3 grid((M + BM - 1) / BM, (N + BN - 1) / BN);

    matmul_2d_tiling << <grid, block >> > (d_a, d_b, d_c, M, N, K, 0, 0, 0, 0);

    hipMemcpy(h_c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void matmul_cublas(float* h_a, float* h_b, float* h_c_cublas, int M, int N, int K) {
    float* d_a, * d_b, * d_c;

    hipMalloc((void**)&d_a, sizeof(float) * M * K);
    hipMalloc((void**)&d_b, sizeof(float) * K * N);
    hipMalloc((void**)&d_c, sizeof(float) * M * N);

    hipMemcpy(d_a, h_a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_b, N, d_a, K, &beta, d_c, N);

    hipMemcpy(h_c_cublas, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void compare_arrays(float* a, float* b, int size) {
    std::map<float, int> error_counts; // To store unique errors and their counts

    for (int i = 0; i < size; i++) {
        double error = std::fabs(a[i] - b[i]);
        if (error > 0.001) { // You may adjust the tolerance level as needed
            error_counts[error]++;
        }
    }

    // Print the unique errors and their counts
    printf("Unique Errors and Their Counts:\n");
    for (const auto& [error, count] : error_counts) {
        printf("Error: %f, Count: %d\n", error, count);
    }
}

int main() {
    const int M = 128;
    const int N = 128;
    const int K = 128;
    const int num_iterations = 1; // Number of iterations for timing

    float* h_a = (float*)malloc(sizeof(float) * M * K);
    float* h_b = (float*)malloc(sizeof(float) * K * N);
    float* h_c = (float*)malloc(sizeof(float) * M * N);
    float* h_c_cublas = (float*)malloc(sizeof(float) * M * N);

    srand(10);
    for (int i = 0; i < M * K; i++) h_a[i] = (float)(rand() % 9);
    for (int i = 0; i < K * N; i++) h_b[i] = (float)(rand() % 9);

    // for (int i = 0; i < M; i++) {
    //     for (int j = 0; j < K; j++) {
    //         printf("%0.2f ", h_a[i * M + j]);
    //     }
    //     printf("\n");
    // }

    // printf("\n");
    // for (int i = 0; i < K; i++) {
    //     for (int j = 0; j < N; j++) {
    //         printf("%0.2f ", h_b[i * K + j]);
    //     }
    //     printf("\n");
    // }

    matmul_cuda(h_a, h_b, h_c, M, N, K);
    matmul_cublas(h_a, h_b, h_c_cublas, M, N, K);

    // printf("\n");
    // for (int i = 0; i < N; i++) {
    //     for (int j = 0; j < M; j++) {
    //         printf("%f ", h_c_cublas[i * N + j]);
    //     }
    //     printf("\n");
    // }
    compare_arrays(h_c, h_c_cublas, M * N);

    // print total error
    float total_error = 0.0f;
    for (int i = 0; i < M * N; i++) {
        total_error += std::abs(h_c[i] - h_c_cublas[i]);
    }
    printf("Total Error: %f\n", total_error);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}

