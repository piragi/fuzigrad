#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "constants.h"

// Declaration of the custom CUDA kernel function (adjust as needed)
extern "C" __global__ void matmul_2d_tiling(float* a, float* b, float* c, const int M, const int N, const int K);

// Custom matrix multiplication function
void matmul(float* a, float* b, float* c, const int M, const int N, const int K) {
    float* d_a, * d_b, * d_c;

    assert(BM == BN);
    assert(((BK * BM) / NUMBER_OF_THREADS) % 4 == 0);

    hipMalloc((void**)&d_a, sizeof(float) * M * K);
    hipMalloc((void**)&d_b, sizeof(float) * K * N);
    hipMalloc((void**)&d_c, sizeof(float) * M * N);

    hipMemcpy(d_a, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    dim3 block(NUMBER_OF_THREADS);
    dim3 grid((M + BM - 1) / BM, (N + BN - 1) / BN);
    matmul_2d_tiling << <grid, block >> > (d_a, d_b, d_c, M, N, K);

    hipMemcpy(c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
