#include "hip/hip_runtime.h"
#include <cstdio>

#include "../constants.h"

__device__ void load_GMEM(float* a, float* a_local, const int idx, const int warp_idx, const int number_of_warps) {
    int global_pos = blockIdx.x * WARPSIZE * number_of_warps * 4;
    int local_pos = ((warp_idx * WARPSIZE) + idx) * 4;
    reinterpret_cast<float4*>(&a_local[local_pos])[0] = reinterpret_cast<float4*>(&a[global_pos + local_pos])[0];
}

__device__ void load_SMEM(float* a_local, float* thread_value, const int idx, const int warp_idx) {
    int local_pos = ((warp_idx * WARPSIZE) + idx) * 4;
    float4 tmp = reinterpret_cast<float4*>(&a_local[local_pos])[0];
    *thread_value += tmp.w + tmp.x + tmp.y + tmp.z;
}

__device__ void shuffle_down_warps_reduce(float* thread_value) {
    unsigned mask = __ballot_sync(0xffffffff, 1);
    for (int offset = WARPSIZE / 2; offset > 0; offset /= 2) {
        *thread_value += __shfl_down_sync(mask, *thread_value, offset);
    }
}

// start block with multiple warps
// do warp reduce
// store result in shared memory
// do shared memory reduce
// store result in global memory
// repeat until only one block left
extern "C" __global__ void reduce_warps(float* a, const int M, float* result) {
    const int idx = threadIdx.x;
    const int warpthread_idx = idx % WARPSIZE;
    const int warp_idx = idx / WARPSIZE;
    const int number_of_warps = REDUCE_NUMBER_OF_THREADS / WARPSIZE;

    float thread_value = 0.0;
    __shared__ float values[WARPSIZE * number_of_warps * 4];

    load_GMEM(a, values, warpthread_idx, warp_idx, number_of_warps);
    __syncthreads();
    load_SMEM(values, &thread_value, warpthread_idx, warp_idx);
    __syncthreads();
    shuffle_down_warps_reduce(&thread_value);
    __syncthreads();

    if (warpthread_idx == 0) {
        int pos = blockIdx.x * number_of_warps + warp_idx;
        result[pos] = thread_value;
    }
}