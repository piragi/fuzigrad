#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>

#include <cstdio>

#include "constants.h"

#define CUDA_CHECK_ERROR(call)                                                                         \
    do {                                                                                               \
        hipError_t err = call;                                                                        \
        if (err != hipSuccess) {                                                                      \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                                        \
        }                                                                                              \
    } while (0)

// Declaration of the custom CUDA kernel function (adjust as needed)
extern "C" __global__ void matmul_2d_tiling(float* a, float* b, float* c, const int M, const int N, const int K);

// Custom matrix multiplication function
extern "C" void matmul(float* a, float* b, float* c, const int M, const int N, const int K) {
    float *d_a, *d_b, *d_c;

    assert(BM == BN);
    assert(((BK * BM) / NUMBER_OF_THREADS) % 4 == 0);
    assert((WM * WN) / (TM * TN * WARPSIZE) >= N_SUBTILES);

    hipMalloc((void**)&d_a, sizeof(float) * M * K);
    hipMalloc((void**)&d_b, sizeof(float) * K * N);
    hipMalloc((void**)&d_c, sizeof(float) * M * N);

    hipMemcpy(d_a, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * K * N, hipMemcpyHostToDevice);

    dim3 block(NUMBER_OF_THREADS);
    dim3 grid((M + BM - 1) / BM, (N + BN - 1) / BN);
    matmul_2d_tiling<<<grid, block>>>(d_a, d_b, d_c, M, N, K);
    CUDA_CHECK_ERROR(hipPeekAtLastError());
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

// Declaration of the custom CUDA kernel function (adjust as needed)
extern "C" __global__ void mean_squared_error(float* a, float* b, float* c, const int M, const int N);

// Custom matrix multiplication function
extern "C" void mse(float* a, float* b, float* c, const int M, const int N) {
    float *d_a, *d_b, *d_c;

    dim3 block(MSE_NUMBER_OF_THREADS);
    dim3 grid((M + MSE_BM - 1) / MSE_BM, (N + MSE_BN - 1) / MSE_BN);
    printf("M: %d, N: %d -- %d blocks and %d threads per block\n", M, N, grid.x * grid.y, block.x);

    hipMalloc((void**)&d_a, sizeof(float) * M * N);
    hipMalloc((void**)&d_b, sizeof(float) * M * N);
    hipMalloc((void**)&d_c, sizeof(float) * grid.x * grid.y);

    hipMemcpy(d_a, a, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * M * N, hipMemcpyHostToDevice);

    mean_squared_error<<<grid, block>>>(d_a, d_b, d_c, M, N);
    CUDA_CHECK_ERROR(hipPeekAtLastError());
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, sizeof(float) * grid.x * grid.y, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

// Declaration of the custom CUDA kernel function (adjust as needed)
extern "C" __global__ void reduce_warps(float* a, const int M, float* result);

// Custom matrix multiplication function
extern "C" void reduce_kernel(float* a, const int M, float* result) {
    float* d_a;
    float* d_result;

    dim3 block(REDUCE_NUMBER_OF_THREADS);
    dim3 grid((M + REDUCE_BM - 1) / REDUCE_BM);
    printf("M: %d -- %d block(s) and %d threads per block\n", M, grid.x * grid.y, block.x);

    hipMalloc((void**)&d_a, sizeof(float) * M);
    hipMalloc((void**)&d_result, sizeof(float) * grid.x);

    hipMemcpy(d_a, a, sizeof(float) * M, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, sizeof(float) * grid.x, hipMemcpyHostToDevice);

    reduce_warps<<<grid, block>>>(d_a, M, d_result);
    CUDA_CHECK_ERROR(hipPeekAtLastError());
    hipDeviceSynchronize();

    hipMemcpy(result, d_result, sizeof(float) * grid.x, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_a);
    hipFree(d_result);
}